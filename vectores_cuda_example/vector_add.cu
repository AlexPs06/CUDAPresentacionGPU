#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define N 1
#define Th 32
#define MAX_ERR 1e-6

__global__ void vector_add(float *out, float *a, float *b, int n) {
   
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < n) out[i] = a[i] + b[i];
}

int main(){
    float *a, *b, *out;
    float *d_a, *d_b, *d_out; 

    // Allocate host memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize host arrays
    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);


    // Allocate device memory
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    dim3 nb( (unsigned int) ceil((double) N/Th) ) ; 
    dim3 nt(Th); 

    // Executing kernel 
    vector_add<<<nb,nt>>>(d_out, d_a, d_b, N);

    // Transfer data back to host memory
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    // Verification
    for(int i = 0; i < N; i++){
        assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }
    printf("out[0] = %f\n", out[0]);
    printf("PASSED\n");


    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("The elapsed time is %f seconds\n", milliseconds/1000);
    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // Deallocate host memory
    free(a); 
    free(b); 
    free(out);
}
