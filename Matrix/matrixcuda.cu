
  
// This program computes a simple version of matrix multiplication
// By: Nick from CoffeeBeforeArch


#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>

using std::cout;
using std::generate;
using std::vector;
using namespace std;
__global__ void matrixMul(const int *a, const int *b, int *c, int N) {
  // Compute each thread's global row and column index
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < N && col < N){
        // Iterate over row, and down column
        c[row * N + col] = 0;
        for (int k = 0; k < N; k++) {
            // Accumulate results for a single element
            c[row * N + col] += a[row * N + k] * b[k * N + col];
        }
    }
  
}

// Check result on the CPU
void verify_result(vector<int> &a, vector<int> &b, vector<int> &c, int N) {
  // For every row...
  for (int i = 0; i < N; i++) {
    // For every column...
    for (int j = 0; j < N; j++) {
      // For every element in the row-column pair
      int tmp = 0;
      for (int k = 0; k < N; k++) {
        // Accumulate the partial results
        tmp += a[i * N + k] * b[k * N + j];
      }
      cout << tmp << "\t";
    //   cout << c[i * N + j] << "\t";
      // Check against the CPU result
      assert(tmp == c[i * N + j]);
    }
    cout << endl;
  }
}

int main() {
  // Matrix size of 1024 x 1024;
  int N = 1024*4;
  // Size (in bytes) of matrix
  size_t bytes = N * N * sizeof(int);

  // Host vectors
  vector<int> h_a(N * N);
  vector<int> h_b(N * N);
  vector<int> h_c(N * N);

//   Initialize matrices
//   generate(h_a.begin(), h_a.end(), []() { return rand() % 100; });
//   generate(h_b.begin(), h_b.end(), []() { return rand() % 100; });

//   Initialize matrices    
    generate(h_a.begin(), h_a.end(), []() { return 1; });
    generate(h_b.begin(), h_b.end(), []() { return 2; });

    // for(int i = 0; i<N * N; i++){
    //         h_a[i]=1;
    //         h_b[i]=2;
    // }
    printf("%i\n",N);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);  
  hipEventRecord(start);

  // Allocate device memory
  int *d_a, *d_b, *d_c;
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);

  // Copy data to the device
  hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);

  // Threads per CTA dimension
  int THREADS = 32;

  // Blocks per grid dimension (assumes THREADS divides N evenly)
  int BLOCKS = ceil (N / THREADS);
  printf("%i\n",BLOCKS);
  if(BLOCKS == 0) BLOCKS++;

  // Use dim3 structs for block  and grid dimensions
  dim3 threads(THREADS, THREADS);
  dim3 blocks(BLOCKS, BLOCKS);

  // Launch kernel
  matrixMul<<<blocks, threads>>>(d_a, d_b, d_c, N);

  // Copy back to the host
  hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost);
  hipEventRecord(stop);
  // Check result
//   verify_result(h_a, h_b, h_c, N);

  cout << "COMPLETED SUCCESSFULLY\n";
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("The elapsed time is %f seconds\n", milliseconds/1000);
  // Free memory on device
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}
